#include "hip/hip_runtime.h"
/* -*- charset: utf-8 -*- */
/*$Id$*/

/* Copyright (c) 2009-2010 Alexey Ozeritsky
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 * 1. Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 * 3. The name of the author may not be used to endorse or promote products
 *    derived from this software without specific prior written permission
 *
 * THIS SOFTWARE IS PROVIDED BY THE AUTHOR ``AS IS'' AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES
 * OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED.
 * IN NO EVENT SHALL THE AUTHOR BE LIABLE FOR ANY DIRECT, INDIRECT,
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT
 * NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
 * DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
 * THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF
 * THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "linal_cuda.h"
#include "texture.h"

register_texture(float, texX1);
register_texture(float, texAX);
register_texture(int, texAP);
register_texture(int, texAI);

namespace linal {

template < typename T, typename AIR, typename AXR, typename XR >
__global__ void ell_mult(
			   T * r, 
			   AIR Ai, 
			   AXR Ax,
			   XR x, 
			   int n,
			   int cols, 
			   int stride)
{
	int row = blockDim.x * blockIdx.x + threadIdx.x;
	if (row < n) {
		T sum = 0;

		for (int i0 = 0; i0 < cols; i0++){
			const T A_ij = Ax.get(stride * i0 + row);

			if (A_ij != 0) {
				const int col = Ai.get(stride * i0 + row);
				sum += A_ij * x.get(col);
			}
		}
	    r[row] = sum;
	}
}

__host__ void 
ell_mult_vector_r(float * r, const int * Ai, const float * Ax, 
	const float * x, int n, int cols, int stride)
{
	SPLAY2(n);

//	texture_reader(texX1) XR(x, n);
//	texture_reader(texAX) AXR(Ax, cols * stride);
//	texture_reader(texAI) AIR(Ai, cols * stride);

	simple_reader < float > XR(x);
	simple_reader < float > AXR(Ax);
	simple_reader < int > AIR(Ai);

	ell_mult<<<blocks, threads>>>(r, AIR, AXR, XR, n, cols, stride);
}

__host__ void 
ell_mult_vector_r(double * r, const int * Ai, const double * Ax, 
	const double * x, int n, int cols, int stride)
{
	SPLAY2(n);

	simple_reader < double > XR(x);
	simple_reader < double > AXR(Ax);
	simple_reader < int > AIR(Ai);
	
	ell_mult<<<blocks, threads>>>(r, AIR, AXR, XR, n, cols, stride);
}

}

