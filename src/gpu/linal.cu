#include "hip/hip_runtime.h"
/* -*- charset: utf-8 -*- */
/*$Id$*/

/* Copyright (c) 2009-2010 Alexey Ozeritsky
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 * 1. Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 * 3. The name of the author may not be used to endorse or promote products
 *    derived from this software without specific prior written permission
 *
 * THIS SOFTWARE IS PROVIDED BY THE AUTHOR ``AS IS'' AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES
 * OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED.
 * IN NO EVENT SHALL THE AUTHOR BE LIABLE FOR ANY DIRECT, INDIRECT,
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT
 * NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
 * DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
 * THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF
 * THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdio.h>

#ifndef WIN32
#include <stdint.h>
#endif

#include "linal_cuda.h"
#include "shmem.h"
#include "texture.h"
#include "reduction.h"

void vector_splay (int n, int threads_min, int threads_max, 
	int grid_width, int *blocks, 
	int *elems_per_block, int * threads_per_block)
{
	if (n < threads_min) {
		*blocks            = 1;
		*elems_per_block   = n;
		*threads_per_block = threads_min;
	} else if (n < (grid_width * threads_min)) {
		*blocks            = ((n + threads_min - 1) / threads_min);
		*threads_per_block = threads_min;
		*elems_per_block   = *threads_per_block;
	} else if (n < (grid_width * threads_max)) {
		int grp;
		*blocks            = grid_width;
		grp                = ((n + threads_min - 1) / threads_min);
		*threads_per_block = (((grp + grid_width -1) / grid_width) * threads_min);
		*elems_per_block   = *threads_per_block;
	} else {
		int grp;
		*blocks            = grid_width;
		*threads_per_block = threads_max;
		grp                = ((n + threads_min - 1) / threads_min);
		grp                = ((grp + grid_width - 1) / grid_width);
		*elems_per_block   = grp * threads_min;
	}
}

//register_texture(float, texX1);
register_texture(float, texAX);
//register_texture(int, texAP);
//register_texture(int, texAI);

register_texture(float, texA);
register_texture(float, texB);

namespace linal {

/* r = k1 * a + k2 * b */
template < typename T >
__global__ void vec_sum1_(T * r, const T * a, const T *b, T k1, T k2, int n)
{
	int threads = gridDim.x  * blockDim.x;
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	for (;i < n; i += threads) {
		r[i] = k1 * a[i] + k2 * b[i];
	}
}

__host__ void vec_sum1(float * r, const float * a, const float *b, float k1, float k2, int n)
{
	SPLAY(n);
	vec_sum1_ <<< blocks, threads >>> (r, a, b, k1, k2, n);
}

__host__ void vec_sum1(double * r, const double * a, const double *b, double k1, double k2, int n)
{
	SPLAY(n);
	vec_sum1_ <<< blocks, threads >>> (r, a, b, k1, k2, n);
}

/* r = a + k2 * b */
template < typename T, typename AR, typename BR >
__global__ void vec_sum2_(T * r, AR a, BR b, T k2, int n)
{
	int threads = gridDim.x  * blockDim.x;
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	for (;i < n; i += threads) {
		r[i] = a.get(i) + k2 * b.get(i);//a[i] + k2 * b[i];
	}
}

__host__ void vec_sum2(float * r, const float * a, const float *b, float k2, int n)
{
	SPLAY2(n);

	bool useTexture;

	useTexture = (n < MAX_1DBUF_SIZE);

	if ((n < 10000) ||
		((!(((uintptr_t) a) % WORD_ALIGN)) && 
		(!(((uintptr_t) b) % WORD_ALIGN)))) 
	{
		useTexture = false;
	}

	if (useTexture) {
		texture_reader(texA) AR(a, n);
		texture_reader(texB) BR(b, n);

		vec_sum2_ <<< blocks, threads >>> (r, AR, BR, k2, n);
	} else {
		simple_reader < float > AR(a);
		simple_reader < float > BR(b);

		vec_sum2_ <<< blocks, threads >>> (r, AR, BR, k2, n);
	}
}

__host__ void vec_sum2(double * r, const double * a, const double *b, double k2, int n)
{
	SPLAY2(n);

	simple_reader < double > AR(a);
	simple_reader < double > BR(b);

	vec_sum2_ <<< blocks, threads >>> (r, AR, BR, k2, n);
}

/* r = a + b */
template < typename T >
__global__ void vec_sum_(T * r, const T * a, const T * b, int n)
{
	int threads = gridDim.x  * blockDim.x;
	int i = blockDim.x * blockIdx.x + threadIdx.x;;
	for (;i < n; i += threads) {
		r[i] = a[i] + b[i];
	}
}

__host__ void vec_sum(float * r, const float * a, const float *b, int n)
{
	SPLAY(n);
	vec_sum_ <<< blocks, threads >>> (r, a, b, n);
}

__host__ void vec_sum(double * r, const double * a, const double *b, int n)
{
	SPLAY(n);
	vec_sum_ <<< blocks, threads >>> (r, a, b, n);
}

/* r = a * b */
template < typename T >
__global__ void vec_mult_(T * r, const T * a, const T * b, int n)
{
	int threads = gridDim.x  * blockDim.x;
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	for (;i < n; i += threads) {
		r[i] = a[i] * b[i];
	}
}

__host__ void vec_mult(float * r, const float * a, const float *b, int n)
{
	SPLAY(n);
	vec_mult_ <<< blocks, threads >>> (r, a, b, n);
}

__host__ void vec_mult(double * r, const double * a, const double *b, int n)
{
	SPLAY(n);
	vec_mult_ <<< blocks, threads >>> (r, a, b, n);
}

/* r = a - b*/
template < typename T >
__global__ void vec_diff_(T * r, const T * a, const T *b, int n)
{
	int threads = gridDim.x  * blockDim.x;
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	for (;i < n; i += threads) {
		r[i] = a[i] - b[i];
	}
}

__host__ void vec_diff(float * r, const float * a, const float *b, int n)
{
	SPLAY(n);
	vec_diff_ <<< blocks, threads >>> (r, a, b, n);
}

__host__ void vec_diff(double * r, const double * a, const double *b,  int n)
{
	SPLAY(n);
	vec_diff_ <<< blocks, threads >>> (r, a, b, n);
}

/* r = b * k*/
template < typename T >
__global__ void vec_mult_scalar_(T * r, const T * b, T k, int n)
{
	int threads = gridDim.x  * blockDim.x;
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	for (;i < n; i += threads) {
		r[i] = k * b[i];
	}
}

__host__ void vec_mult_scalar(float * r, const float * b, float k, int n)
{
	SPLAY2(n);
	vec_mult_scalar_ <<< blocks, threads >>> (r, b, k, n);
}

__host__ void vec_mult_scalar(double * r, const double * b, double k, int n)
{
	SPLAY2(n);
	vec_mult_scalar_ <<< blocks, threads >>> (r, b, k, n);
}

/*
template < typename T >
__global__ void reduction_(T * out, unsigned N, unsigned BlockStride)
{
	unsigned int i      = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int Stride = 2 * BlockStride;
	unsigned int j      = blockDim.x;
	while (j > 0)
	{
		if (Stride * i< N)
			out[Stride*i] += out[Stride*i+ (Stride>>1)];
		Stride <<= 1;
		j >>= 1;
		__syncthreads();
	}
}
*/

#ifdef __DEVICE_EMULATION__
#define EMUSYNC __syncthreads()
#else
#define EMUSYNC
#endif

#include <malloc.h>
#ifdef WIN32
#define alloca _alloca
#endif

template < typename T, typename AR, typename BR >
struct Multiplier
{
	AR a_;
	BR b_;

	Multiplier(AR a, BR b): a_(a), b_(b) {}
	__device__ T get(int i) { 
		return a_.get(i) * b_.get(i);
	}
};

template < typename T >
__host__ T vec_scalar2_(const T * a, const T * b, int n)
{
	int maxThreads = 256;
	int maxBlocks  = 64;

	//int threads = maxThreads;
	//int blocks  = maxBlocks;

	int threads = (n < maxThreads*2) ? nextPow2((n + 1)/ 2) : maxThreads;
	int blocks  = (n + (threads * 2 - 1)) / (threads * 2);
	blocks  = min(maxBlocks, blocks);

	T * v2 = 0;
	T answer = (T)0.0;;

	hipMalloc((void**)&v2, blocks * sizeof(T));

	{
		texture_reader(texA) AR(a, n);
		texture_reader(texB) BR(b, n);
		Multiplier < T, texture_reader(texA), texture_reader(texB) > m(AR, BR);

		//simple_reader < T > AR(a);
		//simple_reader < T > BR(b);
		//Multiplier < T, simple_reader < T >, simple_reader < T > > m(AR, BR);

		reduce6 (threads, blocks, m, v2, n);
	}

	int N = blocks;
	int final_threshold = 1;

	texture_reader(texAX) VR(v2, blocks);
	//simple_reader < T > VR(v2);

	while (N > final_threshold) {
		threads = (N < maxThreads*2) ? nextPow2((N + 1)/ 2) : maxThreads;
		blocks  = (N + (threads * 2 - 1)) / (threads * 2);

		reduce5(threads, blocks, VR, v2, N);

		N = (N + (threads*2-1)) / (threads*2);
	}

	if (final_threshold > 1) {
		T * final = (T*)alloca(N * sizeof(T));
		hipMemcpy(final, v2, N * sizeof(T), hipMemcpyDeviceToHost);
		for (int i = 0; i < N; ++i) 
		{
			answer += final[i];
		}
	} else {
		hipMemcpy(&answer, v2, N * sizeof(T), hipMemcpyDeviceToHost);
	}

	hipFree(v2);

	return answer;
}

/*
__host__ double vec_scalar2(const double * a, const double * b, int n)
{
	return vec_scalar2_(a, b, n);
}
*/

__host__ float vec_scalar2(const float * a, const float * b, int n)
{
	return vec_scalar2_(a, b, n);
}

}
