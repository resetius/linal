#include "hip/hip_runtime.h"
/* -*- charset: utf-8 -*- */
/*$Id$*/

/* Copyright (c) 2009-2010 Alexey Ozeritsky
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 * 1. Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 * 3. The name of the author may not be used to endorse or promote products
 *    derived from this software without specific prior written permission
 *
 * THIS SOFTWARE IS PROVIDED BY THE AUTHOR ``AS IS'' AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES
 * OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED.
 * IN NO EVENT SHALL THE AUTHOR BE LIABLE FOR ANY DIRECT, INDIRECT,
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT
 * NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
 * DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
 * THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF
 * THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "linal_cuda.h"
#include "texture.h"

register_texture(float, texX1);
register_texture(float, texAX);
register_texture(int, texAP);
register_texture(int, texAI);

namespace linal {

template < typename T, typename APR, typename AIR, typename XR, typename AXR >
__global__ void sparse_mult_vector_csr_(T * r, 
	APR Ap, 
	AIR Ai, 
	AXR Ax,
	XR x, 
	int n)
{
	int threads = gridDim.x  * blockDim.x;
	int i, i0, to, j;
	int start = blockDim.x * blockIdx.x + threadIdx.x;

	for (j = start; j < n; j += threads) {
		i0 = Ap.get(j);    
		to = Ap.get(j + 1);

		T rj = (T)0.0;

		for (; i0 < to; ++i0) {
			i   = Ai.get(i0);
			rj += Ax.get(i0) * x.get(i);
		}

		r[j] = rj;
	}
}

__host__ void csr_mult_vector_r(double * r, 
	const int * Ap, 
	const int * Ai, 
	const double * Ax,
	const double * x, 
	int n,
	int nz)
{
	SPLAY(n);
	simple_reader < double > XR(x);
	simple_reader < double > AXR(Ax);
	simple_reader < int > AIR(Ap);
	simple_reader < int > APR(Ap);

	sparse_mult_vector_csr_ <<< blocks, threads >>> (r, APR, AIR, AXR, XR, n);
}

__host__ void csr_mult_vector_r(float * r, 
	const int * Ap, 
	const int * Ai, 
	const float * Ax,
	const float * x, 
	int n,
	int nz)
{
	SPLAY(n);

	bool useTexture;

	useTexture = ((n + 1 < MAX_1DBUF_SIZE) && (nz < MAX_1DBUF_SIZE));

	if (n < 1000 || n > 10000) /* experimental bound */
	{
		useTexture = false;
	}

	if (useTexture) {
		texture_reader(texX1) XR(x, n);
		texture_reader(texAX) AXR(Ax, nz);
		texture_reader(texAI) AIR(Ai, nz);
		texture_reader(texAP) APR(Ap, n + 1);

		sparse_mult_vector_csr_ <<< blocks, threads >>> (r, APR, AIR, AXR, XR, n);
	} else {
		simple_reader < float > XR(x);
		simple_reader < float > AXR(Ax);
		simple_reader < int > AIR(Ai);
		simple_reader < int > APR(Ap);

		sparse_mult_vector_csr_ <<< blocks, threads >>> (r, APR, AIR, AXR, XR, n);
	}
}

}

